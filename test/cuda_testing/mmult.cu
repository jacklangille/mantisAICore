#include <iostream>
#include <hip/hip_runtime.h>
#include <iomanip>

// Kernel function to multiply matrices
__global__ void matrixMul(const float* A, const float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;

    if (row < N && col < N) {
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}


int main() {
    int N = 1024; // Reduced for demonstration purposes
    size_t bytes = N * N * sizeof(float);

    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    // Allocate host memory
    h_A = (float*)malloc(bytes);
    h_B = (float*)malloc(bytes);
    h_C = (float*)malloc(bytes);

    // Initialize matrices on the host
    for (int i = 0; i < N * N; i++) {
        h_A[i] = rand() % 100;
        h_B[i] = rand() % 100;
    }

    // Allocate device memory
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    // Copy data from the host to the device
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    // Number of threads in each thread block
    int blockSize = 64; // A 16x16 thread block size
    dim3 dimBlock(blockSize, blockSize);
    dim3 dimGrid((N + blockSize - 1) / blockSize, (N + blockSize - 1) / blockSize);

    // Create events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Execute the matrix multiplication kernel
    matrixMul<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);

    // Stop timing after kernel execution
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Execution time: " << milliseconds << " ms" << std::endl;

    // Copy the result back to the host
    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
